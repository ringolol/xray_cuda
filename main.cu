#include "hip/hip_runtime.h"
/**
* The CUDA x-ray demo program which calculates an x-ray image.
*
* @author  Valeriy Lyubich
* @version 0.1
* @since   2020-12-18
*/

#include <math.h>
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>

#include "utils.cuh"
#include "matrix.cuh"
#include "beam.cuh"
#include "block.cuh"
#include "f3_overload.cuh"
#include "xray_calc.cuh"
#include "settings.cuh"


/*
    CUDA commands.

    build:
        nvcc -o ./build/app.exe ./main.cu -arch=sm_61
        nvcc -o ./build/cuda_xray.dll --shared ./main.cu -arch=sm_61
    run:
        nvprof ./build/app.exe
        ./build/app.exe
    memory check (build with flags -G and -g):
        cuda-memcheck .\build\app.exe |more
*/


int main() {
    // emulate input
    TubeType tube_type = Be_50;
    float volatage = 100.0;
    float power = 1;
    
    Settings *settings;
    hipMallocManaged(&settings, 1*sizeof(Settings));
    // load common data
    settings->init(tube_type, volatage, power);

    // load materials
    std::vector<float> energy_vec, mean_path_vec;
    read_data("./data/materials/G_Fe.txt", energy_vec, mean_path_vec);
    float *Fe_x;
    hipMallocManaged(&Fe_x, mean_path_vec.size()*sizeof(float));
    copy(mean_path_vec.begin(), mean_path_vec.end(), Fe_x);
    // interpolate mean path for Fe
    int k = 0;
    for(int i = 0; i < 100; i++) {
        while(settings->energy[i] >= energy_vec[k]) {
            k++;
        }
        float mean_path_k = (mean_path_vec[k] - mean_path_vec[k-1]) / (energy_vec[k] - energy_vec[k-1]);
        Fe_x[i] = mean_path_vec[k-1] + (settings->energy[i] - energy_vec[k-1]) * mean_path_k;
        printf("%f %f\n", settings->energy[i], Fe_x[i]);
    }

    // x-ray source
    float3 source = make_float3(0.0, 0.0, 0.0);
    // blocks representing 3d objects
    Block* blocks;
    // sensor matrix
    Matrix* matrix;

    // allocate memorry in managed memory
    int blocks_num = 6;
    hipMallocManaged(&blocks, blocks_num*sizeof(Block));
    hipMallocManaged(&matrix, 1*sizeof(Matrix));
    float3 *block1_points, *block2_points, *block3_points, *block4_points, *block5_points, *block6_points;
    hipMallocManaged(&block1_points, 4*sizeof(float3));
    hipMallocManaged(&block2_points, 4*sizeof(float3));
    hipMallocManaged(&block3_points, 4*sizeof(float3));
    hipMallocManaged(&block4_points, 4*sizeof(float3));
    hipMallocManaged(&block5_points, 4*sizeof(float3));
    hipMallocManaged(&block6_points, 4*sizeof(float3));

    // init blocks
    float p_hsize = 70; //4.5 (edge case)
    float p_z = -35;
    float p1_thicc = 2, p2_thicc = 2;
    float hole_size = 1;
    float hh = hole_size/2;

    // first layer
    block1_points[0] = make_float3(-p_hsize, -p_hsize, p_z);
    block1_points[1] = make_float3( p_hsize, -p_hsize, p_z);
    block1_points[2] = make_float3(-p_hsize,  p_hsize, p_z);
    block1_points[3] = make_float3(-p_hsize, -p_hsize, p_z-p1_thicc);
    blocks[0].init(block1_points, iron, Fe_x);

    // second layer with the hole
    block2_points[0] = make_float3(-p_hsize, -p_hsize, p_z-p1_thicc);
    block2_points[1] = make_float3(-hh,      -p_hsize, p_z-p1_thicc);
    block2_points[2] = make_float3(-p_hsize, p_hsize,  p_z-p1_thicc);
    block2_points[3] = make_float3(-p_hsize, -p_hsize, p_z-p1_thicc-hole_size);
    blocks[1].init(block2_points, iron, Fe_x);

    block3_points[0] = make_float3(hh,      -p_hsize, p_z-p1_thicc);
    block3_points[1] = make_float3(p_hsize, -p_hsize, p_z-p1_thicc);
    block3_points[2] = make_float3(hh,      p_hsize,  p_z-p1_thicc);
    block3_points[3] = make_float3(hh,      -p_hsize, p_z-p1_thicc-hole_size);
    blocks[2].init(block3_points, iron, Fe_x);

    block4_points[0] = make_float3(-hh, -p_hsize, p_z-p1_thicc);
    block4_points[1] = make_float3(hh,  -p_hsize, p_z-p1_thicc);
    block4_points[2] = make_float3(-hh, -hh,      p_z-p1_thicc);
    block4_points[3] = make_float3(-hh, -p_hsize, p_z-p1_thicc-hole_size);
    blocks[3].init(block4_points, iron, Fe_x);

    block5_points[0] = make_float3(-hh, hh,      p_z-p1_thicc);
    block5_points[1] = make_float3(hh,  hh,      p_z-p1_thicc);
    block5_points[2] = make_float3(-hh, p_hsize, p_z-p1_thicc);
    block5_points[3] = make_float3(-hh, hh,      p_z-p1_thicc-hole_size);
    blocks[4].init(block5_points, iron, Fe_x);

    // third layer
    block6_points[0] = make_float3(-p_hsize, -p_hsize, p_z-p1_thicc-hole_size);
    block6_points[1] = make_float3( p_hsize, -p_hsize, p_z-p1_thicc-hole_size);
    block6_points[2] = make_float3(-p_hsize,  p_hsize, p_z-p1_thicc-hole_size);
    block6_points[3] = make_float3(-p_hsize, -p_hsize, p_z-p1_thicc-hole_size-p2_thicc);
    blocks[5].init(block6_points, iron, Fe_x);

    // init matrix
    float matrix_width = 40;
    int matrix_width_px = 1024;
    int matrix_height_px = 1024;
    matrix->init(matrix_width_px, matrix_height_px, matrix_width/matrix_width_px, -90.0);
    printf("matrix size: %dx%d\n", matrix->width, matrix->height);

    // start x-ray image calculation
    int threads_size = 32;
    dim3 threadsPerBlock(threads_size, threads_size);
    int blocks_width = ceil((float)matrix_width_px/threads_size);
    int blocks_height = ceil((float)matrix_height_px/threads_size);
    dim3 blocksShape(blocks_width, blocks_height);

    // run kernel to calculate x-ray image
    xray_image_kernel<<<blocksShape, threadsPerBlock>>>(source, blocks, blocks_num, matrix);

    // wait for all threads and blocks
    hipDeviceSynchronize();

    std::ofstream outdata("output.txt");
    for(int i = 0; i < matrix->width; i++) {
        for(int j = 0; j < matrix->height; j++) {
            outdata << matrix->image[i][j];
            if(j != matrix->height-1)
                outdata << '\t';
        }
        if(i != matrix->width-1)
            outdata << '\n';
    }
    outdata.close();

    // print errors
    gpuErrchk( hipPeekAtLastError() );

    // free allocated memory
    hipFree(blocks);
    hipFree(matrix);
    hipFree(block1_points);
    hipFree(block2_points);

    hipProfilerStop();
    return 0;
}